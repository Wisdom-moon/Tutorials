/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/* Vector addition: C = A + B.
 *
 * This sample is a very basic sample that implements element by element
 * vector addition. It is the same as the sample illustrating Chapter 3
 * of the programming guide with some additions like error checking.
 *
 */

// Includes
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

// Variables
float* h_A;
float* h_B;
float* h_C;
float* d_A;
float* d_B;
float* d_C;
bool noprompt = false;

const bool pin_generic_memory = false;

// Functions
void CleanupResources(void);
void RandomInit(float*, int);
void ParseArguments(int, char**);

// Device code
__global__ void VecAdd(const float* A, const float* B, float* C, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N)
        C[i] = A[i] + B[i];
}

// Host code
int main(int argc, char** argv)
{
    float elapsed_time, time_memcpy, time_kernel;   // timing variables
    int  device_sync_method = hipDeviceScheduleBlockingSync; // by default we use BlockingSync

    int N = 65536;

    if (argc > 1)
      N = atoi(argv[1]);

    size_t size = N * sizeof(float);

    //printf("Vector Addition with %d elements.\n", N);
    hipSetDevice( 0 );
    //cudaSetDeviceFlags( device_sync_method );
    hipSetDeviceFlags(hipDeviceMapHost);

    // Allocate input vectors h_A and h_B in host memory
  if (pin_generic_memory)
  {
    h_A = (float *) malloc(size);
    h_B = (float *) malloc(size);
    h_C = (float *) malloc(size);
    (hipHostRegister(h_A, size, hipHostMallocMapped));
    (hipHostRegister(h_B, size, hipHostMallocMapped));
    (hipHostRegister(h_C, size, hipHostMallocMapped));
  }
  else
  {
    (hipHostAlloc((void **)&h_A, size, hipHostMallocMapped));
    (hipHostAlloc((void **)&h_B, size, hipHostMallocMapped));
    (hipHostAlloc((void **)&h_C, size, hipHostMallocMapped));
  }

    // Initialize input vectors
    RandomInit(h_A, N);
    RandomInit(h_B, N);

    // allocate and initialize an array of stream handles
    //Create CUDA event to calculate time.
    hipEvent_t start_event, stop_event;
    int eventflags = hipEventBlockingSync;
    hipEventCreateWithFlags(&start_event, eventflags);
    hipEventCreateWithFlags(&stop_event, eventflags);

    hipHostGetDevicePointer((void **)&d_A, (void *)h_A, 0);
    hipHostGetDevicePointer((void **)&d_B, (void *)h_B, 0);
    hipHostGetDevicePointer((void **)&d_C, (void *)h_C, 0);
    // Allocate vectors in device memory

    // Copy vectors from host memory to device memory
    hipEventRecord(start_event, 0);     // record in stream-0, to ensure that all previous CUDA calls have completed

    // Invoke kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    VecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
    //cudaDeviceSynchronize();

    hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event);   // block until the event is actually recorded
    hipEventElapsedTime(&time_memcpy, start_event, stop_event);
    printf("%f\n", time_memcpy);
    
    // Verify result
    /*
    int i;
    for (i = 0; i < N; ++i) {
        float sum = h_A[i] + h_B[i];
        if (fabs(h_C[i] - sum) > 1e-5)
            break;
    }

    
    if ( i < N)
	printf("Wrong Value: %dth element!\n", i);
    else
	printf("Passed!\n");
	*/
    

    // release resources
    hipEventDestroy(start_event);
    hipEventDestroy(stop_event);
    CleanupResources();

    return 0;
}

void CleanupResources(void)
{
  if (pin_generic_memory)
  {
    hipHostUnregister(h_A);
    hipHostUnregister(h_B);
    hipHostUnregister(h_C);
    free(h_A);
    free(h_B);
    free(h_C);
  }
  else
  {
    // Free host memory
    if (h_A)
        hipHostFree(h_A);
    if (h_B)
        hipHostFree(h_B);
    if (h_C)
        hipHostFree(h_C);
  }
        
    hipDeviceReset();
}

// Allocates an array with random float entries.
void RandomInit(float* data, int n)
{
    for (int i = 0; i < n; ++i)
        data[i] = rand() / (float)RAND_MAX;
}

// Parse program arguments
void ParseArguments(int argc, char** argv)
{
    for (int i = 0; i < argc; ++i)
        if (strcmp(argv[i], "--noprompt") == 0 ||
			strcmp(argv[i], "-noprompt") == 0) 
		{
            noprompt = true;
            break;
        }
}
