#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 *
 *
 * This sample illustrates the usage of CUDA streams for overlapping
 * kernel execution with device/host memcopies.  The kernel is used to 
 * initialize an array to a specific value, after which the array is 
 * copied to the host (CPU) memory.  To increase performance, multiple 
 * kernel/memcopy pairs are launched asynchronously, each pair in its 
 * own stream.  Devices with Compute Capability 1.1 can overlap a kernel
 * and a memcopy as long as they are issued in different streams.  Kernels
 * are serialized.  Thus, if n pairs are launched, streamed approach
 * can reduce the memcopy cost to the (1/n)th of a single copy of the entire
 * data set.
 *
 * Additionally, this sample uses CUDA events to measure elapsed time for
 * CUDA calls.  Events are a part of CUDA API and provide a system independent
 * way to measure execution times on CUDA devices with approximately 0.5 
 * microsecond precision.
 *
 * Elapsed times are averaged over nreps repetitions (10 by default).
 *
*/

const char *sSDKsample = "simpleStreams";

const char *sEventSyncMethod[] = 
{ 
	"hipEventDefault", 
	"hipEventBlockingSync", 
	"hipEventDisableTiming", 
	NULL 
};

const char *sDeviceSyncMethod[] = 
{ 
	"hipDeviceScheduleAuto", 
	"hipDeviceScheduleSpin", 
	"hipDeviceScheduleYield", 
	"INVALID", 
	"hipDeviceScheduleBlockingSync", 
	NULL 
};

// CUDA utilities and system includes
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

// Include headers
#include <stdio.h>
#include <stdlib.h>

#ifndef WIN32
#include <sys/mman.h> // for mmap() / munmap()
#endif

__global__ void init_array(int *g_data, int *factor, int num_iterations)
{ 
   int idx = blockIdx.x * blockDim.x + threadIdx.x;

    for(int i=0;i<num_iterations;i++)
	    g_data[idx] += *factor;	// non-coalesced on purpose, to burn time
}

int correct_data(int *a, const int n, const int c)
{
    for(int i = 0; i < n; i++) {
        if(a[i] != c) {
           printf("%d: %d %d\n", i, a[i], c);
           return 0;
        }
    }
    return 1;
}

void AllocateHostMemory(bool bPinGenericMemory, int **pp_a, int nbytes)
{
#if CUDART_VERSION >= 4000
    if (bPinGenericMemory)
    {
        // allocate a generic page-aligned chunk of system memory
    #ifdef WIN32
        printf("> VirtualAlloc() allocating %4.2f Mbytes of (generic page-aligned system memory)\n", (float)nbytes/1048576.0f);
        *pp_a = (int *) VirtualAlloc( NULL, nbytes, MEM_RESERVE|MEM_COMMIT, PAGE_READWRITE );
    #else
        printf("> mmap() allocating %4.2f Mbytes (generic page-aligned system memory)\n", (float)nbytes/1048576.0f);
        *pp_a = (int *) mmap( NULL, nbytes, PROT_READ|PROT_WRITE, MAP_PRIVATE|MAP_ANON, -1, 0 );
    #endif
        // pin allocate memory
        ( hipHostRegister(*pp_a, nbytes, hipHostRegisterMapped) );
    }
    else
#endif
    {
        printf("> hipHostMalloc() allocating %4.2f Mbytes of system memory\n", (float)nbytes/1048576.0f);
        // allocate host memory (pinned is required for achieve asynchronicity)
        ( hipHostMalloc((void**)pp_a, nbytes) ); 
    }
}

void FreeHostMemory(bool bPinGenericMemory, int **pp_a, int nbytes)
{
#if CUDART_VERSION >= 4000
    // CUDA 4.0 support pinning of generic host memory
    if (bPinGenericMemory)
    {
        // unpin and delete host memory
        ( hipHostUnregister(*pp_a) );
  #ifdef WIN32
        VirtualFree(*pp_a, 0, MEM_RELEASE);
  #else
        munmap(*pp_a, nbytes);
  #endif
    }
    else
#endif
    {
        hipHostFree(*pp_a);
    }
}

void printHelp()
{
    printf("Usage: %s [options below]\n", sSDKsample);
    printf("\t--sync_method for CPU/GPU synchronization\n");
    printf("\t             (0=Automatic Blocking Scheduling)\n");
    printf("\t             (1=Spin Blocking Scheduling)\n");
    printf("\t             (2=Yield Blocking Scheduling)\n");
    printf("\t   <Default> (4=Blocking Sync Event Scheduling for low CPU utilization)\n");
    printf("\t--use_generic_memory (default) use generic page-aligned for system memory\n");
    printf("\t--use_cuda_malloc_host (optional) use hipHostMalloc to allocate system memory\n");
}

#if defined(__APPLE__) || defined(MACOSX)
#define DEFAULT_PINNED_GENERIC_MEMORY false
#else
#define DEFAULT_PINNED_GENERIC_MEMORY true
#endif

int main(int argc, char *argv[])
{
    int cuda_device = 0;
    int nstreams = 2;               // number of streams for CUDA calls
    int nreps = 10;                 // number of times each experiment is repeated
    int n = 16 * 1024 * 1024;       // number of ints in the data set
    int nbytes = n * sizeof(int);   // number of data bytes
    dim3 threads, blocks;           // kernel launch configuration
    float elapsed_time, time_memcpy, time_kernel;   // timing variables
    float scale_factor = 1.0f;

    // allocate generic memory and pin it laster instead of using hipHostAlloc()

    bool bPinGenericMemory  = DEFAULT_PINNED_GENERIC_MEMORY; // we want this to be the default behavior
    int  device_sync_method = hipDeviceScheduleBlockingSync; // by default we use BlockingSync

    int niterations;	// number of iterations for the loop inside the kernel


    printf("[ %s ]\n\n", sSDKsample);
    printf("\n> ");
    cuda_device = 0;

    // check the compute capability of the device
    int num_devices=0;
    ( hipGetDeviceCount(&num_devices) );
    if(0==num_devices)
    {
        printf("your system does not have a CUDA capable device, waiving test...\n");
	return -1;
    }
	
    // check if the command-line chosen device ID is within range, exit if not
    if( cuda_device >= num_devices )
    {
        printf("cuda_device=%d is invalid, must choose device ID between 0 and %d\n", cuda_device, num_devices-1);
	return -1;
    }

    hipSetDevice( cuda_device );

    hipDeviceProp_t deviceProp;
    ( hipGetDeviceProperties(&deviceProp, cuda_device) );
    if( (1 == deviceProp.major) && (deviceProp.minor < 1)) {
        printf("%s does not have Compute Capability 1.1 or newer.  Reducing workload.\n", deviceProp.name);
    }

    if(deviceProp.major >= 2) {
        niterations = 100;
    } else {		    
        if(deviceProp.minor > 1) {
            niterations = 5;
        } else {
            niterations = 1; // reduced workload for compute capability 1.0 and 1.1
        }
    }

#ifdef __DEVICE_EMULATION__
    n = 4096;   // reduced workload for emulation (n should be divisible by 512*nstreams)
#else
    // Anything that is less than 32 Cores will have scaled down workload
    scale_factor = max((32.0f / (72 * (float)deviceProp.multiProcessorCount) ), 1.0f);
    n = (int)rint( (float)n / scale_factor );
#endif

    printf("> CUDA Capable: SM %d.%d hardware\n", deviceProp.major, deviceProp.minor);
    printf("> %d Multiprocessor(s) x %d (Cores/Multiprocessor) = %d (Cores)\n", 
            deviceProp.multiProcessorCount,
            72,
            72 * deviceProp.multiProcessorCount);

    printf("> scale_factor = %1.4f\n", 1.0f/scale_factor);
    printf("> array_size   = %d\n\n", n);

    // enable use of blocking sync, to reduce CPU usage
    printf("> Using CPU/GPU Device Synchronization method (%s)\n", sDeviceSyncMethod[device_sync_method]);
    hipSetDeviceFlags( device_sync_method | (bPinGenericMemory ? hipDeviceMapHost : 0 ) );

    // allocate host memory
    int c = 5;                      // value to which the array will be initialized
    int *h_a = 0;                     // pointer to the array data in host memory

    // Allocate Host memory (could be using hipHostMalloc or VirtualAlloc/mmap if using the new CUDA 4.0 features
    AllocateHostMemory(bPinGenericMemory, &h_a, nbytes);

    // allocate device memory
    int *d_a = 0, *d_c = 0;             // pointers to data and init value in the device memory
    ( hipMalloc((void**)&d_a, nbytes) );
    ( hipMalloc((void**)&d_c, sizeof(int)) );
    ( hipMemcpy(d_c, &c, sizeof(int), hipMemcpyHostToDevice) );

    printf("\nStarting Test\n");

    // allocate and initialize an array of stream handles
    hipStream_t *streams = (hipStream_t*) malloc(nstreams * sizeof(hipStream_t));
    for(int i = 0; i < nstreams; i++) {
        ( hipStreamCreate(&(streams[i])) );
    }

    // create CUDA event handles
    // use blocking sync
    hipEvent_t start_event, stop_event;
    int eventflags = ( (device_sync_method == hipDeviceScheduleBlockingSync) ? hipEventBlockingSync: hipEventDefault );

    ( hipEventCreateWithFlags(&start_event, eventflags) );
    ( hipEventCreateWithFlags(&stop_event, eventflags) );

    // time memcopy from device
    hipEventRecord(start_event, 0);     // record in stream-0, to ensure that all previous CUDA calls have completed
    hipMemcpyAsync(h_a, d_a, nbytes, hipMemcpyDeviceToHost, streams[0]);
    hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event);   // block until the event is actually recorded
    ( hipEventElapsedTime(&time_memcpy, start_event, stop_event) );
    printf("memcopy:\t%.2f\n", time_memcpy);
    
    // time kernel
    threads=dim3(512, 1);
    blocks=dim3(n / threads.x, 1);
    hipEventRecord(start_event, 0);
    init_array<<<blocks, threads, 0, streams[0]>>>(d_a, d_c, niterations);
    hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event);
    ( hipEventElapsedTime(&time_kernel, start_event, stop_event) );
    printf("kernel:\t\t%.2f\n", time_kernel);

    //////////////////////////////////////////////////////////////////////
    // time non-streamed execution for reference
    threads=dim3(512, 1);
    blocks=dim3(n / threads.x, 1);
    hipEventRecord(start_event, 0);
    for(int k = 0; k < nreps; k++)
    {
        init_array<<<blocks, threads>>>(d_a, d_c, niterations);
        hipMemcpy(h_a, d_a, nbytes, hipMemcpyDeviceToHost);
    }
    hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event);
    ( hipEventElapsedTime(&elapsed_time, start_event, stop_event) );
    printf("non-streamed:\t%.2f (%.2f expected)\n", elapsed_time / nreps, time_kernel + time_memcpy);

    //////////////////////////////////////////////////////////////////////
    // time execution with nstreams streams
    threads=dim3(512,1);
    blocks=dim3(n/(nstreams*threads.x),1);
    memset(h_a, 255, nbytes);     // set host memory bits to all 1s, for testing correctness
    hipMemset(d_a, 0, nbytes); // set device memory to all 0s, for testing correctness
    hipEventRecord(start_event, 0);
    for(int k = 0; k < nreps; k++)
    {
        // asynchronously launch nstreams kernels, each operating on its own portion of data
        for(int i = 0; i < nstreams; i++)
            init_array<<<blocks, threads, 0, streams[i]>>>(d_a + i * n / nstreams, d_c, niterations);

        // asynchronously launch nstreams memcopies.  Note that memcopy in stream x will only
        //   commence executing when all previous CUDA calls in stream x have completed
        for(int i = 0; i < nstreams; i++)
            hipMemcpyAsync(h_a + i * n / nstreams, d_a + i * n / nstreams, nbytes / nstreams, hipMemcpyDeviceToHost, streams[i]);
    }
    hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event);
    ( hipEventElapsedTime(&elapsed_time, start_event, stop_event) );
    printf("%d streams:\t%.2f (%.2f expected with compute capability 1.1 or later)\n", nstreams, elapsed_time / nreps, time_kernel + time_memcpy / nstreams);

    // check whether the output is correct
    printf("-------------------------------\n");
    bool bResults = (correct_data(h_a, n, c*nreps*niterations));

    // release resources
    for(int i = 0; i < nstreams; i++) {
        hipStreamDestroy(streams[i]);
    }
    hipEventDestroy(start_event);
    hipEventDestroy(stop_event);

    // Free hipHostMalloc or Generic Host allocated memory (from CUDA 4.0)
    FreeHostMemory( bPinGenericMemory, &h_a, nbytes );

    hipFree(d_a);
    hipFree(d_c);

    hipDeviceReset();
    return 0;
}
