#include "hip/hip_runtime.h"
/**
 * This version is stamped on May 10, 2016
 *
 * Contact:
 *   Louis-Noel Pouchet <pouchet.ohio-state.edu>
 *   Tomofumi Yuki <tomofumi.yuki.fr>
 *
 * Web address: http://polybench.sourceforge.net
 */
/* covariance.c: this file is part of PolyBench/C */

#include <stdio.h>
#include <unistd.h>
#include <string.h>
#include <math.h>
/* Array initialization. */
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

/* Include polybench common header. */
#include <polybench.h>

/* Include benchmark-specific header. */
#include "covariance.h"

int threadsPerBlock = 2;



__global__ void my_kernel ( int m, int n, double float_n, double (*cov)[1200] , double (*data)[1200] , int length, int offset)
{

  int i;
  int j;
  int k;
  i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i >= length)
    return;
  i += offset;

    for (j = i; j < _PB_M; j++)
      {
        cov[i][j] = SCALAR_VAL(0.0);
        for (k = 0; k < _PB_N; k++)
	  cov[i][j] += data[k][i] * data[k][j];
        cov[i][j] /= (float_n - SCALAR_VAL(1.0));
        cov[j][i] = cov[i][j];
      }
}

static
void init_array (int m, int n,
		 DATA_TYPE *float_n,
		 DATA_TYPE POLYBENCH_2D(data,N,M,n,m))
{
  int i, j;

  *float_n = (DATA_TYPE)n;

  for (i = 0; i < N; i++)
    for (j = 0; j < M; j++)
      data[i][j] = ((DATA_TYPE) i*j) / M;
}


/* DCE code. Must scan the entire live-out data.
   Can be used also to check the correctness of the output. */
static
void print_array(int m,
		 DATA_TYPE POLYBENCH_2D(cov,M,M,m,m))

{
  int i, j;

  POLYBENCH_DUMP_START;
  POLYBENCH_DUMP_BEGIN("cov");
  for (i = 0; i < m; i++)
    for (j = 0; j < m; j++) {
      if ((i * m + j) % 20 == 0) fprintf (POLYBENCH_DUMP_TARGET, "\n");
      fprintf (POLYBENCH_DUMP_TARGET, DATA_PRINTF_MODIFIER, cov[i][j]);
    }
  POLYBENCH_DUMP_END("cov");
  POLYBENCH_DUMP_FINISH;
}


/* Main computational kernel. The whole function will be timed,
   including the call and return. */
static
void kernel_covariance(int m, int n,
		       DATA_TYPE float_n,
		       DATA_TYPE POLYBENCH_2D(data,N,M,n,m),
		       DATA_TYPE POLYBENCH_2D(cov,M,M,m,m),
		       DATA_TYPE POLYBENCH_1D(mean,M,m))
{
  hipSetDevice(0);
  hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
  hipStream_t *streams = (hipStream_t*) malloc(nstreams*sizeof(hipStream_t));
  for (int i = 0; i < nstreams; i++) {
    hipStreamCreate(&(streams[i]));
  }

  hipEvent_t start_event, stop_event;
  float time_elapsed;
  int eventflags = hipEventBlockingSync;
  hipEventCreateWithFlags(&start_event, eventflags);
  hipEventCreateWithFlags(&stop_event, eventflags);
  printf("%d\t%d\t%d\t", (((m)-1)-0 + 1), threadsPerBlock, nstreams);
  double (*d_cov)[1200];
  hipMalloc((void **)&d_cov, (((m)-1)+ 1)* sizeof (double [1200]));
  double (*d_data)[1200];
  hipMalloc((void **)&d_data, (((n)-1)+ 1)* sizeof (double [1200]));
  int i, j, k;

  for (j = 0; j < _PB_M; j++)
    {
      mean[j] = SCALAR_VAL(0.0);
      for (i = 0; i < _PB_N; i++)
        mean[j] += data[i][j];
      mean[j] /= float_n;
    }

  for (i = 0; i < _PB_N; i++)
    for (j = 0; j < _PB_M; j++)
      data[i][j] -= mean[j];


    int ni = m;
    int totalblocks = (ni +threadsPerBlock - 1) / threadsPerBlock;
    int refblocks = totalblocks % nstreams;
    int idlethreads = ni % threadsPerBlock ? threadsPerBlock - ni % threadsPerBlock : 0;
    int blocksPerGrid[nstreams];
    int threadsPerSubtask[nstreams];
    int offset[nstreams];
    for (int i = 0; i < nstreams; i++)
    {
      blocksPerGrid[i] = totalblocks / nstreams;
      if (i < refblocks)
        blocksPerGrid[i] ++;
      threadsPerSubtask[i] = threadsPerBlock * blocksPerGrid[i];
    }
    threadsPerSubtask[nstreams - 1] -= idlethreads;
    offset[0] = 0;
    for (int i = 1; i < nstreams; i++)
      offset[i] = offset[i-1] + threadsPerSubtask[i-1];

hipEventRecord(start_event, 0);
hipMemcpyAsync(d_data, data, (((n)-1)+ 1)* sizeof (double [1200]), hipMemcpyHostToDevice, streams[0]);
for (int i = 0; i < nstreams; i++)
{
  my_kernel<<<blocksPerGrid[i], threadsPerBlock,0, streams[i]>>>(m, n, float_n, d_cov, d_data, threadsPerSubtask[i], offset[i]);
}
hipMemcpyAsync(cov, d_cov, (((m)-1)+ 1)* sizeof (double [1200]),hipMemcpyDeviceToHost, streams[0]);
hipEventRecord(stop_event, 0);
hipEventSynchronize(stop_event);
hipEventElapsedTime(&time_elapsed, start_event, stop_event);
printf("%f\n", time_elapsed);

}


int main(int argc, char** argv)
{
  /* Retrieve problem size. */
  int n = N;
  int m = M;

  if (argc > 1) 
    threadsPerBlock  = atoi(argv[1]);
  if (argc > 2)
    nstreams = atoi(argv[2]);

  /* Variable declaration/allocation. */
  DATA_TYPE float_n;
  POLYBENCH_2D_ARRAY_DECL(data,DATA_TYPE,N,M,n,m);
  POLYBENCH_2D_ARRAY_DECL(cov,DATA_TYPE,M,M,m,m);
  POLYBENCH_1D_ARRAY_DECL(mean,DATA_TYPE,M,m);


  /* Initialize array(s). */
  init_array (m, n, &float_n, POLYBENCH_ARRAY(data));

  /* Start timer. */
  polybench_start_instruments;

  /* Run kernel. */
  kernel_covariance (m, n, float_n,
		     POLYBENCH_ARRAY(data),
		     POLYBENCH_ARRAY(cov),
		     POLYBENCH_ARRAY(mean));

  /* Stop and print timer. */
  polybench_stop_instruments;
  polybench_print_instruments;

  /* Prevent dead-code elimination. All live-out data must be printed
     by the function call in argument. */
  polybench_prevent_dce(print_array(m, POLYBENCH_ARRAY(cov)));

  /* Be clean. */
  POLYBENCH_FREE_ARRAY(data);
  POLYBENCH_FREE_ARRAY(cov);
  POLYBENCH_FREE_ARRAY(mean);

  return 0;
}

